#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <time.h>
#include ""
#include <iostream>
#include <io.h>
#include <stdlib.h>



// includes CUDA
#include <hip/hip_runtime_api.h>

// includes, project


// d is the number of float in the input alphabet 

#define MOD 1000000007
#define M  3
#define N  13

__global__ void STRING(int *txt_d, int *pat_d, int *f_d, int hp, int ht, int val, int random)
{

	int id = blockIdx.x*blockDim.x + threadIdx.x;


	int j, i;

	if (id <= N - M){


		if (hp == ht){
			int j;

			for (j = 0; j < M; j++){
				if (txt_d[id + j] != pat_d[j])
					break;
			}

			if (j == M)
				f_d[id] = id;
			//found in text
		}
		
	}
}
/* Driver program to test above function */
	int main(int argc, int ** argv)
	{


		int *txt;
		int *pat;

		clock_t start, end;
		float run_time;
		//float txt[], pat[]; // pointers to host memory; (CPU)
		int *txt_d, *pat_d, number; // pointers to device memory; (GPU)
		int i, j, index;
		int  *f_d, *f;
		txt = (int *)malloc(sizeof(int)*N);
		pat = (int *)malloc(sizeof(int)*M);
		f = (int *)malloc(sizeof(int)*(N - M));

		FILE *fp;
		fp = fopen("text.txt", "rt");
		if (fp == NULL)
		{
			printf("cannot open file \ n");
			getchar();
			exit(1);
		}
		int temp4;
		for (i = 0; i < N; i++)
		{
			fscanf(fp, "%d", &temp4);
			txt[i] = temp4;
		}
		fclose(fp);
		//FILE *fp;
		fp = fopen("pat.txt", "rt");
		if (fp == NULL)
		{
			printf("cannot open file \ n");
			getchar();
			exit(1);
		}
		int temp5;
		for (i = 0; i < M; i++)

		{
			fscanf(fp, "%d", &temp5);
			pat[i] = temp5;
		}
		fclose(fp);
		
	

		// allocate arrays on device
		hipMalloc((void **)&txt_d, N*sizeof(int));
		hipMalloc((void **)&pat_d, M*sizeof(int));
		hipMalloc((void **)&f_d, (N - M)*sizeof(int));

		//dim3 dimBlock(blocksize, blocksize);
		//dim3 dimGrid(ceil(float(n) / float(dimBlock.x)), ceil(float(n) / float(dimBlock.y)));
		//////// copy and run the code on the device


		int hp = 0, ht = 0, val = 1;

		//srand(time(0));
		int random = rand() % (MOD - 1) + 1; // generating random value x

		for (int i = 0; i < M; i++){
			hp = (random*hp) % MOD; // calculating hash of pattern
			ht = (random*ht) % MOD; // calculating hash of first sub-string
			// of text
			hp += pat[i];
			ht += txt[i];

			hp %= MOD;
			ht %= MOD;
			val = (val*random) % MOD;
		}




		hipMemcpy(txt_d, txt, N*sizeof(int), hipMemcpyHostToDevice);

		hipMemcpy(pat_d, pat, M*sizeof(int), hipMemcpyHostToDevice);

		/*dim3 dimGrid((N-M,1));
		dim3 dimBlock(M,1);*/
		start = clock();


		STRING << <1,10 >> >(txt_d, pat_d, f_d, hp, ht, val, random);
		hipDeviceSynchronize();
		end = clock();

		hipMemcpy(f, f_d, (N - M)*sizeof(int), hipMemcpyDeviceToHost);

		for (int j = 0; j < N; j++){
			printf("txt[%d]=%d \n", j, txt[j]);
		}
		for (int j = 0; j < M; j++){
			printf("pat[%d]=%d \n", j, pat[j]);
		}
		for (int j = 0; j < N - M; j++){
			printf("Pattern found at index f[%d]=%d \n", j, f[j]);
		}
		//printf("%s \n", txt);
		run_time = (float(end - start)) / CLOCKS_PER_SEC;
		printf("\n\ntime=%f", run_time);
		free(txt);
		free(pat);
		free(f);
		hipFree(txt_d);
		hipFree(pat_d);
		hipFree(f_d);

		getchar();
		return 0;
	}
